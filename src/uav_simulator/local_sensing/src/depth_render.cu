#include "hip/hip_runtime.h"
#include "depth_render.cuh"
__global__ void render(float3 *data_devptr, Parameter *para_devptr, DeviceImage<int> *depth_devptr)
{
	const int index = threadIdx.x + blockIdx.x * blockDim.x;
	const Parameter para = *para_devptr;
	if(index >= para.point_number)
		return;
	float3 my_point = data_devptr[index];
	
	//transform
	float3 trans_point;
	trans_point.x = my_point.x * para.r[0][0] + my_point.y * para.r[0][1] + my_point.z * para.r[0][2] + para.t[0];
	trans_point.y = my_point.x * para.r[1][0] + my_point.y * para.r[1][1] + my_point.z * para.r[1][2] + para.t[1];
	trans_point.z = my_point.x * para.r[2][0] + my_point.y * para.r[2][1] + my_point.z * para.r[2][2] + para.t[2];

	if(trans_point.z <= 0.0f)
		return;

	//project
	int2 projected;
	projected.x = trans_point.x / trans_point.z * para.fx + para.cx + 0.5;
	projected.y = trans_point.y / trans_point.z * para.fy + para.cy + 0.5;
	if(projected.x < 0 || projected.x >= para.width || projected.y < 0 || projected.y >= para.height)
		return;

	// float dist = length(trans_point);
	float dist = trans_point.z;
	int dist_mm = dist * 1000.0f + 0.5f;

	//int r = 0.0173 * para.fx / dist + 0.5f;
//	int r = 0.0473 * para.fx / dist + 0.5f;
	int r = 0.0573 * para.fx / dist + 0.5f;
	for(int i = -r; i <= r; i++)
		for(int j = -r; j <= r; j++)
		{
			int to_x = projected.x + j;
			int to_y = projected.y + i;
			if(to_x < 0 || to_x >= para.width || to_y < 0 || to_y >= para.height)
				continue;
			int *dist_ptr = &(depth_devptr->atXY(to_x, to_y));
			atomicMin(dist_ptr, dist_mm);
		}
}

__global__ void depth_initial(DeviceImage<int> *depth_devptr)
{
  const int x = threadIdx.x + blockIdx.x * blockDim.x;
  const int y = threadIdx.y + blockIdx.y * blockDim.y;
  int width = depth_devptr->width;
  int height = depth_devptr->height;

  if(x >= width || y >= height)
  	return;

  depth_devptr->atXY(x,y) = 999999;
}

DepthRender::DepthRender():
	cloud_size(0),
	host_cloud_ptr(NULL),
	dev_cloud_ptr(NULL),
	has_devptr(false)
{
}

DepthRender::~DepthRender()
{
	if(has_devptr)
	{
		free(host_cloud_ptr);
		hipFree(dev_cloud_ptr);
		hipFree(parameter_devptr);
	}
}

void DepthRender::set_para(float _fx, float _fy, float _cx, float _cy, int _width, int _height)
{
	parameter.fx = _fx;
	parameter.fy = _fy;
	parameter.cx = _cx;
	parameter.cy = _cy;
	parameter.width = _width;
	parameter.height = _height;
}

void DepthRender::set_data(vector<float> &cloud_data)
{
	cloud_size = cloud_data.size() / 3;
	parameter.point_number = cloud_size;

	host_cloud_ptr = (float3*) malloc(cloud_size * sizeof(float3));
	for(int i = 0; i < cloud_size; i++)
		host_cloud_ptr[i] = make_float3(cloud_data[3*i], cloud_data[3*i+1], cloud_data[3*i+2]);

  hipError_t err = hipMalloc(&dev_cloud_ptr, cloud_size * sizeof(float3));
  if(err != hipSuccess)
    throw CudaException("DeviceLinear: unable to allocate linear memory.", err);
 	err = hipMemcpy(
          dev_cloud_ptr,
          host_cloud_ptr,
          cloud_size * sizeof(float3),
          hipMemcpyHostToDevice);
  if(err != hipSuccess)
  	throw CudaException("DeviceLinear: unable to copy data from host to device.", err);

  err = hipMalloc(&parameter_devptr, sizeof(Parameter));
  if(err != hipSuccess)
    throw CudaException("DeviceLinear: unable to allocate linear memory.", err);
 	err = hipMemcpy(
          parameter_devptr,
          &parameter,
          sizeof(Parameter),
          hipMemcpyHostToDevice);
  if(err != hipSuccess)
  	throw CudaException("DeviceLinear: unable to copy data from host to device.", err);

  has_devptr = true;

  //printf("load points done!\n");
}

/*void DepthRender::render_pose( Matrix3d &rotation, Vector3d &translation, int *host_ptr)
{
	for(int i = 0; i < 3; i++)
	{
		parameter.t[i] = translation(i);
		for(int j = 0; j < 3; j++)
		{
			parameter.r[i][j] = rotation(i,j);
		}
	}
 	hipError_t err = hipMemcpy(
          parameter_devptr,
          &parameter,
          sizeof(Parameter),
          hipMemcpyHostToDevice);
  if(err != hipSuccess)
  	throw CudaException("DeviceLinear: unable to copy data from host to device.", err);

	DeviceImage<int> depth_output(parameter.width, parameter.height);
  depth_output.zero();

  dim3 depth_block;
  dim3 depth_grid;
  depth_block.x = 16;
  depth_block.y = 16;
  depth_grid.x = (parameter.width + depth_block.x - 1 ) / depth_block.x;
  depth_grid.y = (parameter.height + depth_block.y - 1 ) / depth_block.y;
  depth_initial<<<depth_grid, depth_block>>>(depth_output.dev_ptr);

  dim3 render_block;
  dim3 render_grid;
  render_block.x = 64;
  render_grid.x = (cloud_size + render_block.x - 1) / render_block.x;
  render<<<render_grid, render_block>>>(dev_cloud_ptr, parameter_devptr, depth_output.dev_ptr);

	depth_output.getDevData(host_ptr);
}
*/
//void DepthRender::render_pose( Matrix4d &transformation, int *host_ptr)
void DepthRender::render_pose( double * transformation, int *host_ptr)
{
	for(int i = 0; i < 3; i++)
	{
		parameter.t[i] = transformation[4 * i + 3];//transformation(i,3);
		for(int j = 0; j < 3; j++)
		{
			parameter.r[i][j] = transformation[4 * i + j];//transformation(i,j);
		}
	}
 	hipError_t err = hipMemcpy(
          parameter_devptr,
          &parameter,
          sizeof(Parameter),
          hipMemcpyHostToDevice);
  if(err != hipSuccess)
  	throw CudaException("DeviceLinear: unable to copy data from host to device.", err);

	DeviceImage<int> depth_output(parameter.width, parameter.height);
  depth_output.zero();

  dim3 depth_block;
  dim3 depth_grid;
  depth_block.x = 16;
  depth_block.y = 16;
  depth_grid.x = (parameter.width + depth_block.x - 1 ) / depth_block.x;
  depth_grid.y = (parameter.height + depth_block.y - 1 ) / depth_block.y;
  depth_initial<<<depth_grid, depth_block>>>(depth_output.dev_ptr);

  dim3 render_block;
  dim3 render_grid;
  render_block.x = 64;
  render_grid.x = (cloud_size + render_block.x - 1) / render_block.x;
  render<<<render_grid, render_block>>>(dev_cloud_ptr, parameter_devptr, depth_output.dev_ptr);

	depth_output.getDevData(host_ptr);
}